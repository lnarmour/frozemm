#include "hip/hip_runtime.h"
#include "matmultKernel.h"
#include <stdio.h>

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){
  float *Asub, *Bsub, *Csub;
  int num_threads=BLOCK_SIZE_X*BLOCK_SIZE_Y;
  Csub = &C.elements[C.stride * STRIP_SIZE * blockIdx.y + FOOTPRINT_SIZE_X * blockIdx.x];
  
  float Cvalues[SCALING_FACTOR] = {[0 ... SCALING_FACTOR-1]=0};
  
  // Read each strip of A and B from global DRAM into shared memory
  // perform matrix product of strips and accumate into Cvalues
  for (int m = 0;  m < (A.width / STRIP_SIZE); ++m){ 
    Asub = &A.elements[A.stride * STRIP_SIZE * m + FOOTPRINT_SIZE_Y * blockIdx.y];
    Bsub = &B.elements[B.stride * STRIP_SIZE * m + FOOTPRINT_SIZE_X * blockIdx.x];
    
    __shared__ float shared_A[STRIP_SIZE][FOOTPRINT_SIZE_Y];
    __shared__ float shared_B[STRIP_SIZE][FOOTPRINT_SIZE_X];
   
    // transpose the TT strip of Asub into SS strip in shared_A
    for (int i=threadIdx.y; i<STRIP_SIZE; i+=BLOCK_SIZE_Y)
      for (int j=threadIdx.x; j<FOOTPRINT_SIZE_X; j+=BLOCK_SIZE_X){
        shared_A[i][j] = Asub[i*A.stride + j];
        shared_B[i][j] = Bsub[i*B.stride + j];
        if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0 && m == 0) {
          printf("m=%d   shared_A[%d][%d] = %f\n", m, i, j, shared_A[i][j]);
        }
    }
    __syncthreads();
    
    int c=0;
    for (int i=0; i<STRIP_SIZE; i=i+(SCALING_FACTOR_Y*num_threads))
      for (int j=0; j<FOOTPRINT_SIZE_X; j=j+(SCALING_FACTOR_X*num_threads))
        for (int l=0; l<SCALING_FACTOR_X; ++l)
          for (int k=0; k<SCALING_FACTOR_Y; ++k, c++)
            for (int e=0; e <STRIP_SIZE; ++e)
              Cvalues[c] += shared_A[e][threadIdx.y*SCALING_FACTOR_Y + i + l] * shared_B[e][threadIdx.x*SCALING_FACTOR_X + j + k];

    __syncthreads();
  }


  // Write Cvalues back to global DRAM
  int d=0;
  for(int i=0; i<STRIP_SIZE; i=i+(SCALING_FACTOR_Y*num_threads))
    for (int j=0; j<FOOTPRINT_SIZE_X; j=j+(SCALING_FACTOR_X*num_threads))
      for (int l=0; l<SCALING_FACTOR_X;++l)
        for (int k=0; k<SCALING_FACTOR_Y;++k)
          Csub[(threadIdx.y*SCALING_FACTOR_Y + i + l)*C.stride+(threadIdx.x*SCALING_FACTOR_X+j)+k]=Cvalues[d++];
}
