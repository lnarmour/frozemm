// taken from:
// https://developer.nvidia.com/sites/default/files/akamai/cuda/files/Misc/mygpu.pdf

// nvcc 036 sgemm .c -lcublas
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "nvmlPower.hpp"

#define IDX(i,j,ld) (((j)*(ld))+(i))
#define RUNS 1

int checkt(long, long, float*, float*, float*);

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

int main (int argc, char** argv) {
  long M,N,K;
  if (argc>1) {
    M = N = K = atol(argv[1]);
  } else {
    printf("Usage: %s N\n", argv[0]);
    return 1;
  }

  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  long i,j; 
  float* A;
  float* B;
  float* C;
  A = (float*)malloc(M*N*sizeof(float)); 
  B = (float*)malloc(M*N*sizeof(float)); 
  C = (float*)malloc(M*N*sizeof(float)); 

  for(j=0; j<N; j++) 
    for(i=0; i<M; i++) 
      A[IDX(i,j,M)] = (float) ((i*j+1) % M) / M;

  // on the device
  float * d_A;
  float * d_B;
  float * d_C;
  cudaStat = hipMalloc((void**)&d_A, M*N*sizeof(*A));
  cudaStat = hipMalloc((void**)&d_B, M*N*sizeof(*B));
  cudaStat = hipMalloc((void**)&d_C, M*N*sizeof(*C));
  stat = hipblasCreate(&handle); // initialize CUBLAS context

  // copy matrices from the host to the device
  stat = hipblasSetMatrix(M, N, sizeof(*A), A, M, d_A, M); //A -> d_A
  stat = hipblasSetMatrix(M, N, sizeof(*B), B, M, d_B, M); //B -> d_B
  stat = hipblasSetMatrix(M, N, sizeof(*C), C, M, d_C, M); //C -> d_C
  float alpha = 1.0;
  float beta = 0.0;
  
#ifndef CHECK
  // Invoke kernel for warm up
  stat = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, &alpha, d_A, M, &beta, d_B, M, d_C, M);
#endif

  // Synchronize to make sure everyone is done in the warmup.
  hipDeviceSynchronize();

  // events for timing
  hipEvent_t startEvent, stopEvent;
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  float ms;

  // ------------
  // time kernel
  // ------------

  nvmlAPIRun();
  checkCuda( hipEventRecord(startEvent, 0) );
  for (int r=0; r<RUNS; r++)
    stat = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, &alpha, d_A, M, &beta, d_B, M, d_C, M);
  checkCuda( hipEventRecord(stopEvent, 0) );
  hipDeviceSynchronize();
  nvmlAPIEnd();

  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );

  double time = ms/1000;

  printf( "Time: %lf (sec)\n", time);

  stat = hipblasGetMatrix(M, N, sizeof(*C), d_C, M, C, M); //d_C -> C
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipblasDestroy(handle);

  #ifdef CHECK
  int ret = checkt(M,N,A,B,C);
  #endif

  free (A);
  free (B);
  free (C);
  return EXIT_SUCCESS ;
}
